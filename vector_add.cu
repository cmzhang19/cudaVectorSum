#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include "vector_add.h"

__global__ void vector_add(int *a, int *b, int *c, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if( i < n )
    {
        c[i] = a[i] + b[i];
        //printf("%d\n", c[i]);
    }
}


void add_vectors(int *a, int *b, int *c, int n)
{
    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, n * sizeof(int));
    hipMalloc(&d_b, n * sizeof(int));
    hipMalloc(&d_c, n * sizeof(int));

    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, n * sizeof(int), hipMemcpyHostToDevice);

    const int threadPerBlock = 256;
    const int blockPerGrid = (n + threadPerBlock - 1) / threadPerBlock;

    vector_add<<<blockPerGrid, threadPerBlock>>>(d_a, d_b, d_c, n);

    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
